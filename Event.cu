#include "hip/hip_runtime.h"
# include "Event.cuh"


const unsigned int N_MAX_HITS_PER_MODULE = 100;
const unsigned int N_MAX_MD_PER_MODULES = 100;
const unsigned int N_MAX_SEGMENTS_PER_MODULE = 600; //WHY!
const unsigned int MAX_CONNECTED_MODULES = 40;
const unsigned int N_MAX_TRACKLETS_PER_MODULE = 5000;//temporary
const unsigned int N_MAX_TRIPLETS_PER_MODULE = 1000;
struct SDL::modules* SDL::modulesInGPU = nullptr;
struct SDL::modules* SDL::modulesInHost = nullptr;//explicit
unsigned int SDL::nModules;

SDL::Event::Event()
{
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    mdsInTemp = nullptr; //explicit
    segmentsInGPU = nullptr;
    segmentsInTemp = nullptr;
    trackletsInGPU = nullptr; 
    trackletsInTemp = nullptr; 
    tripletsInGPU = nullptr; 
    tripletsInTemp = nullptr; 
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_tracklets_by_layer_barrel_[i] = 0;
        n_triplets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
	    n_segments_by_layer_endcap_[i] = 0;
            n_tracklets_by_layer_endcap_[i] = 0;
            n_triplets_by_layer_endcap_[i] = 0;
        }
    }
    resetObjectsInModule();

}

SDL::Event::~Event()
{

#ifdef Explicit_MD
    mdsInTemp->freeMemory();
    hipFree(mdsInTemp);
#else
    mdsInGPU->freeMemory();
#endif
#ifdef Explicit_Seg
    segmentsInTemp->freeMemory(); 
    hipFree(segmentsInTemp);
#else
    segmentsInGPU->freeMemory(); 
#endif
#ifdef Explicit_Tracklet
    trackletsInTemp->freeMemory(); 
    hipFree(trackletsInTemp);
#else
    trackletsInGPU->freeMemory();
#endif
#ifdef Explicit_Trips
    tripletsInTemp->freeMemory(); 
    hipFree(tripletsInTemp);
#else
    tripletsInGPU->freeMemory();
#endif
    hitsInGPU->freeMemory();
    hipFree(hitsInGPU);
    hipFree(mdsInGPU);
    hipFree(segmentsInGPU);
    hipFree(trackletsInGPU);
    hipFree(tripletsInGPU);
}

void SDL::initModules()
{
    hipMallocManaged(&modulesInGPU, sizeof(struct SDL::modules));
    if((modulesInGPU->detIds) == nullptr) //check for nullptr and create memory
    {
        loadModulesFromFile(*modulesInGPU,nModules); //nModules gets filled here
    }
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::Event::addHitToEvent(float x, float y, float z, unsigned int detId)
{
    const int HIT_MAX = 1000000;
    const int HIT_2S_MAX = 100000;

    if(hitsInGPU == nullptr)
    {
        hipMallocManaged(&hitsInGPU, sizeof(SDL::hits));
        createHitsInUnifiedMemory(*hitsInGPU,HIT_MAX,HIT_2S_MAX);
    }

    //calls the addHitToMemory function
    addHitToMemory(*hitsInGPU, *modulesInGPU, x, y, z, detId);

    unsigned int moduleLayer = modulesInGPU->layers[(*detIdToIndex)[detId]];
    unsigned int subdet = modulesInGPU->subdets[(*detIdToIndex)[detId]];

    if(subdet == Barrel)
    {
        n_hits_by_layer_barrel_[moduleLayer-1]++;
    }
    else
    {
        n_hits_by_layer_endcap_[moduleLayer-1]++;
    } 

}

void SDL::Event::addMiniDoubletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(mdsInGPU->nMDs[idx] == 0 or modulesInGPU->hitRanges[idx * 2] == -1)
        {
            modulesInGPU->mdRanges[idx * 2] = -1;
            modulesInGPU->mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            modulesInGPU->mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + mdsInGPU->nMDs[idx] - 1;
    
            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[idx] -1] += mdsInGPU->nMDs[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += mdsInGPU->nMDs[idx];
            }

        }
    }
}

void SDL::Event::addSegmentsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(segmentsInGPU->nSegments[idx] == 0)
        {
            modulesInGPU->segmentRanges[idx * 2] = -1;
            modulesInGPU->segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            modulesInGPU->segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU->nSegments[idx] - 1;

            //for(unsigned int jdx = 0; jdx < segmentsInGPU->nSegments[idx]; jdx++)
            //    printSegment(*segmentsInGPU, *mdsInGPU, *hitsInGPU, *modulesInGPU, idx * N_MAX_SEGMENTS_PER_MODULE + jdx);

            if(modulesInGPU->subdets[idx] == Barrel)
            {
  
                n_segments_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += segmentsInGPU->nSegments[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[idx] -1] += segmentsInGPU->nSegments[idx];
            }
        }
    }
}

void SDL::Event::createMiniDoublets()
{
    hipDeviceSynchronize();
    auto memStart = std::chrono::high_resolution_clock::now();
    if(mdsInGPU == nullptr)
    {
        hipMallocManaged(&mdsInGPU, sizeof(SDL::miniDoublets)); // use for unified memory and explicit to get transfer back
        //hipMalloc(&mdsInGPU, sizeof(SDL::miniDoublets)); //completely explicit. issues with transfer back. use this version
#ifdef Explicit_MD
      hipHostMalloc(&mdsInTemp, sizeof(SDL::miniDoublets));
    	createMDsInExplicitMemory(*mdsInGPU,*mdsInTemp, N_MAX_MD_PER_MODULES, nModules);
#else
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules);
#endif
    }
#ifdef Explicit_MD
    hipMemset(mdsInGPU->nMDs,0,nModules*sizeof(unsigned int));
#endif
    hipDeviceSynchronize();
    auto memStop = std::chrono::high_resolution_clock::now();
    auto memDuration = std::chrono::duration_cast<std::chrono::milliseconds>(memStop - memStart); //in milliseconds

    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1),(N_MAX_HITS_PER_MODULE % nThreads.y == 0 ? N_MAX_HITS_PER_MODULE/nThreads.y : N_MAX_HITS_PER_MODULE/nThreads.y + 1), (N_MAX_HITS_PER_MODULE % nThreads.z == 0 ? N_MAX_HITS_PER_MODULE/nThreads.z : N_MAX_HITS_PER_MODULE/nThreads.z + 1));
    //std::cout<<nBlocks.x<<" "<<nBlocks.y<<" "<<nBlocks.z<<" "<<std::endl;

//    int nThreads = 1;
//    int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    hipDeviceSynchronize();
    auto syncStart = std::chrono::high_resolution_clock::now();

    createMiniDoubletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    auto syncStop = std::chrono::high_resolution_clock::now();

    auto syncDuration =  std::chrono::duration_cast<std::chrono::milliseconds>(syncStop - syncStart);

    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    
    }
    
#ifdef AddObjects
    addMiniDoubletsToEvent();
#endif


}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        hipMallocManaged(&segmentsInGPU, sizeof(SDL::segments));
        //hipMalloc(&segmentsInGPU, sizeof(SDL::segments));

#ifdef Explicit_Seg
        hipHostMalloc(&segmentsInTemp, sizeof(SDL::segments));
        createSegmentsInExplicitMemory(*segmentsInGPU, *segmentsInTemp,N_MAX_SEGMENTS_PER_MODULE, nModules);
#else
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules);
#endif
    }
#ifdef Explicit_Seg
    hipMemset(segmentsInGPU->nSegments,0,nModules*sizeof(unsigned int));
#endif
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

//    dim3 nThreads(1,16,16);
//    dim3 nBlocks(((nLowerModules * MAX_CONNECTED_MODULES)  % nThreads.x == 0 ? (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x : (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x + 1),(N_MAX_MD_PER_MODULES % nThreads.y == 0 ? N_MAX_MD_PER_MODULES/nThreads.y : N_MAX_MD_PER_MODULES/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));

    unsigned int nThreads = 1;   
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createSegmentsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    
    }
#ifdef AddObjects
    addSegmentsToEvent();
#endif

}


void SDL::Event::createTriplets()
{
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    if(tripletsInGPU == nullptr)
    {
        //hipMalloc(&tripletsInGPU, sizeof(SDL::triplets));
        hipMallocManaged(&tripletsInGPU, sizeof(SDL::triplets));
#ifdef Explicit_Trips
        hipHostMalloc(&tripletsInTemp,sizeof(SDL::triplets));
        createTripletsInExplicitMemory(*tripletsInGPU,*tripletsInTemp, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#else
        createTripletsInUnifiedMemory(*tripletsInGPU, N_MAX_TRIPLETS_PER_MODULE, nLowerModules);
#endif
    }
#ifdef Explicit_Trips
    hipMemset(tripletsInGPU->nTriplets,0,nLowerModules*sizeof(unsigned int));
#endif

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;
    createTripletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *tripletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;
    }
#ifdef AddObjects
    addTripletsToEvent();
#endif
}


void SDL::Event::createTrackletsWithModuleMap()
{
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        //hipMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
        hipMallocManaged(&trackletsInGPU, sizeof(SDL::tracklets));

#ifdef Explicit_Tracklet
        hipHostMalloc(&trackletsInTemp,sizeof(SDL::tracklets));
        createTrackletsInExplicitMemory(*trackletsInGPU,*trackletsInTemp, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#endif
    }
#ifdef Explicit_Tracklet
    hipMemset(trackletsInGPU->nTracklets,0,nLowerModules*sizeof(unsigned int));
#endif

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    

    }
    /*addTrackletsToEvent will be called in the createTrackletsWithAGapWithModuleMap function*/

#ifdef AddObjects
    addTrackletsToEvent();
#endif

}


void SDL::Event::createTrackletsWithAGapWithModuleMap()
{
    //use the same trackletsInGPU as before if it exists
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        //hipMalloc(&trackletsInGPU, sizeof(SDL::tracklets));
        hipMallocManaged(&trackletsInGPU, sizeof(SDL::tracklets));

#ifdef Explicit_Tracklet
        hipHostMalloc(&trackletsInTemp,sizeof(SDL::tracklets));
        createTrackletsInExplicitMemory(*trackletsInGPU,*trackletsInTemp, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#else
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
#endif
    }
#ifdef Explicit_Tracklet
    hipMemset(trackletsInGPU->nTracklets,0,nLowerModules*sizeof(unsigned int));
#endif

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsWithAGapInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    

    }

}


void SDL::Event::addTrackletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(trackletsInGPU->nTracklets[i] == 0)
        {
            modulesInGPU->trackletRanges[idx * 2] = -1;
            modulesInGPU->trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            modulesInGPU->trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + trackletsInGPU->nTracklets[i] - 1;

            //for(unsigned int jdx = 0; jdx < trackletsInGPU->nTracklets[i]; jdx++)
            //{
            //    printTracklet(*trackletsInGPU, *segmentsInGPU, *mdsInGPU, *hitsInGPU, *modulesInGPU, i * N_MAX_TRACKLETS_PER_MODULE + jdx);
            //}
            
 
            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
        }
    }
}

__global__ void testMiniDoublets(struct SDL::miniDoublets& mdsInGPU){
  int tid = threadIdx.x+(blockDim.x*blockIdx.x);
    if(tid<2000){
    printf("testmini %d: %d\n",tid,mdsInGPU.nMDs[tid]);
  }
}

void SDL::Event::addTripletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(tripletsInGPU->nTriplets[i] == 0)
        {
            modulesInGPU->tripletRanges[idx * 2] = -1;
            modulesInGPU->tripletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->tripletRanges[idx * 2] = idx * N_MAX_TRIPLETS_PER_MODULE;
            modulesInGPU->tripletRanges[idx * 2 + 1] = idx * N_MAX_TRIPLETS_PER_MODULE + tripletsInGPU->nTriplets[i] - 1;
 
            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_triplets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
            else
            {
                n_triplets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += tripletsInGPU->nTriplets[i];
            }
        }
    }
}
__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;
    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
    
    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;

        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
    }
}

/*__global__ void createMiniDoubletsFromLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int lowerModuleIndex, unsigned int upperModuleIndex, unsigned int nLowerHits, unsigned int nUpperHits)
{
    unsigned int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);

    if(success)
    {
        unsigned int mdModuleIdx = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        unsigned int mdIdx = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIdx;

        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIdx);
    }
}


__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;

    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(1,nLowerHits % nThreads.y == 0 ? nLowerHits/nThreads.y : nLowerHits/nThreads.y + 1, nUpperHits % nThreads.z == 0 ? nUpperHits/nThreads.z : nUpperHits/nThreads.z + 1);

    createMiniDoubletsFromLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, lowerModuleIndex, upperModuleIndex, nLowerHits, nUpperHits);

  
}*/

/*__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int xAxisIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int innerMDArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int outerMDArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;

    int innerLowerModuleArrayIdx = xAxisIdx/MAX_CONNECTED_MODULES;
    int outerLowerModuleArrayIdx = xAxisIdx % MAX_CONNECTED_MODULES; //need this index from the connected module array
    
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIdx];

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];

    if(outerLowerModuleArrayIdx >= nConnectedModules) return;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIdx];

    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex];
    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex];

    if(innerMDArrayIdx >= nInnerMDs) return;
    if(outerMDArrayIdx >= nOuterMDs) return;

    unsigned int innerMDIndex = modulesInGPU.mdRanges[innerLowerModuleIndex * 2] + innerMDArrayIdx;
    unsigned int outerMDIndex = modulesInGPU.mdRanges[outerLowerModuleIndex * 2] + outerMDArrayIdx;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;

        addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
    }


}*/


__global__ void createSegmentsFromInnerLowerModule(struct SDL::modules&modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerLowerModuleIndex, unsigned int nInnerMDs)
{
    unsigned int outerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerMDArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int outerMDArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIndex];

    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex];
    if(innerMDArrayIndex >= nInnerMDs) return;
    if(outerMDArrayIndex >= nOuterMDs) return;

    unsigned int innerMDIndex = innerLowerModuleIndex * N_MAX_MD_PER_MODULES + innerMDArrayIndex;
    unsigned int outerMDIndex = outerLowerModuleIndex * N_MAX_MD_PER_MODULES + outerMDArrayIndex;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;

        addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
    }

}

__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int innerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIndex];
    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];
    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex];

    if(nConnectedModules == 0) return;

    if(nInnerMDs == 0) return;
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nConnectedModules % nThreads.x == 0 ? nConnectedModules/nThreads.x : nConnectedModules/nThreads.x + 1), (nInnerMDs % nThreads.y == 0 ? nInnerMDs/nThreads.y : nInnerMDs/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));
    createSegmentsFromInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerLowerModuleIndex,nInnerMDs);
   
}


__global__ void createTrackletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;
        //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;


    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //number of possible outer segment inner MD lower modules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }

}

__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Proposal 1 : Inner kernel takes care of both loops
    int xAxisIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex =  blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
    
    if(innerSegmentArrayIndex >= nInnerSegments) return;

    int middleLowerModuleArrayIndex = xAxisIndex / MAX_CONNECTED_MODULES;
    int outerInnerLowerModuleArrayIndex = xAxisIndex % MAX_CONNECTED_MODULES;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex]; 
   
    //first check for middle modules
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    //second check for outerInnerLowerMoules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }
}

__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 1
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) % nThreads.x  == 0 ? (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x : (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

/*__global__ void createTrackletsWithAGapFromMiddleLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int middleLowerModuleIndex, unsigned int innerSegmentIndex, unsigned int innerInnerLowerModuleArrayIndex, unsigned int nOuterInnerLowerModules,unsigned int innerOuterLowerModuleIndex)
{
    //Inner kernel of Proposal 2 : Inner kernel does middle->outer modoule mapping
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    
    //check for outerInnerLowerModules
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;


    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex and innerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }
}

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Middle kernel of Proposal 2 : middle kernel does the inner->middle module mapping

    int middleLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - modules that are connected to outer lower module of inner segment
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
 
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;
 
    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];

    dim3 nThreads(1,32,1);
    dim3 nBlocks(nOuterInnerLowerModules % nThreads.x  == 0 ? nOuterInnerLowerModules / nThreads.x : nOuterInnerLowerModules / nThreads.x + 1 ,N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.y : N_MAX_SEGMENTS_PER_MODULE/nThreads.y + 1,1);

    createTrackletsWithAGapFromMiddleLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex, middleLowerModuleIndex,innerSegmentIndex,innerInnerLowerModuleArrayIndex,nOuterInnerLowerModules,innerOuterLowerModuleIndex);


}


__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 2
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,1,1);
    dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 , nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,1);
    
    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, innerInnerLowerModuleArrayIndex);

}*/

__global__ void createTripletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int nConnectedModules, unsigned int innerInnerLowerModuleArrayIndex)
{
    int innerSegmentArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - outer lower module of inner segment
    unsigned int middleLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[middleLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;
    unsigned int outerSegmentIndex = middleLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

    bool success = runTripletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut);
    if(success)
    {
        unsigned int tripletModuleIndex = atomicAdd(&tripletsInGPU.nTriplets[innerInnerLowerModuleArrayIndex], 1);
        unsigned int tripletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRIPLETS_PER_MODULE + tripletModuleIndex;

        addTripletToMemory(tripletsInGPU, innerSegmentIndex, outerSegmentIndex, innerInnerLowerModuleIndex, middleLowerModuleIndex, outerOuterLowerModuleIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, tripletIndex, betaInCut, betaOutCut, dBetaCut);
    }   
}

__global__ void createTripletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::triplets& tripletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerInnerLowerModuleIndex];
    if(nConnectedModules == 0) return;

    dim3 nThreads(16,16,1);
    dim3 nBlocks(nInnerSegments % nThreads.x == 0 ? nInnerSegments / nThreads.x : nInnerSegments / nThreads.x + 1, N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE / nThreads.y : N_MAX_SEGMENTS_PER_MODULE / nThreads.y + 1);

    createTripletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, tripletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, nConnectedModules, innerInnerLowerModuleArrayIndex);
}
unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;
   
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];   
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;
   
}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];   
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTracklets()
{
    unsigned int tracklets = 0;
    for(auto &it:n_tracklets_by_layer_barrel_)
    {
        tracklets += it;
    }
    for(auto &it:n_tracklets_by_layer_endcap_)
    {
        tracklets += it;
    }

    return tracklets;
   
}

unsigned int SDL::Event::getNumberOfTrackletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_tracklets_by_layer_barrel_[layer];
    else
        return n_tracklets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerBarrel(unsigned int layer)
{
    return n_tracklets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerEndcap(unsigned int layer)
{
    return n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTriplets()
{
    unsigned int triplets = 0;
    for(auto &it:n_triplets_by_layer_barrel_)
    {
        triplets += it;
    }
    for(auto &it:n_triplets_by_layer_endcap_)
    {
        triplets += it;
    }

    return triplets;
   
}


unsigned int SDL::Event::getNumberOfTripletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_triplets_by_layer_barrel_[layer];
    else
        return n_triplets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerBarrel(unsigned int layer)
{
    return n_triplets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTripletsByLayerEndcap(unsigned int layer)
{
    return n_triplets_by_layer_endcap_[layer];
}

struct SDL::hits* SDL::Event::getHits()
{
    return hitsInGPU;
}

struct SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}

struct SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}

struct SDL::tracklets* SDL::Event::getTracklets()
{
    return trackletsInGPU;
}

struct SDL::triplets* SDL::Event::getTriplets()
{
    return tripletsInGPU;
}
