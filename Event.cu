#include "hip/hip_runtime.h"
# include "Event.cuh"


const unsigned int N_MAX_HITS_PER_MODULE = 100;
const unsigned int N_MAX_MD_PER_MODULES = 100;
const unsigned int N_MAX_SEGMENTS_PER_MODULE = 600; //WHY!
const unsigned int MAX_CONNECTED_MODULES = 40;
const unsigned int N_MAX_TRACKLETS_PER_MODULE = 5000;//temporary
struct SDL::modules* SDL::modulesInGPU = nullptr;
unsigned int SDL::nModules;

SDL::Event::Event()
{
    hitsInGPU = nullptr;
    mdsInGPU = nullptr;
    segmentsInGPU = nullptr;
    trackletsInGPU = nullptr; 
    //reset the arrays
    for(int i = 0; i<6; i++)
    {
        n_hits_by_layer_barrel_[i] = 0;
        n_minidoublets_by_layer_barrel_[i] = 0;
        n_segments_by_layer_barrel_[i] = 0;
        n_tracklets_by_layer_barrel_[i] = 0;
        if(i<5)
        {
            n_hits_by_layer_endcap_[i] = 0;
            n_minidoublets_by_layer_endcap_[i] = 0;
	        n_segments_by_layer_endcap_[i] = 0;
            n_tracklets_by_layer_endcap_[i] = 0;
        }
    }
    resetObjectsInModule();

}

SDL::Event::~Event()
{
    hitsInGPU->freeMemory();
    hipFree(hitsInGPU);
    mdsInGPU->freeMemory();
    hipFree(mdsInGPU);
    segmentsInGPU->freeMemory(); 
    hipFree(segmentsInGPU);
    trackletsInGPU->freeMemory();
    hipFree(trackletsInGPU);
}

void SDL::initModules()
{
    hipMallocManaged(&modulesInGPU, sizeof(struct SDL::modules));
    if((modulesInGPU->detIds) == nullptr) //check for nullptr and create memory
    {
        loadModulesFromFile(*modulesInGPU,nModules); //nModules gets filled here
    }
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::Event::resetObjectsInModule()
{
    resetObjectRanges(*modulesInGPU,nModules);
}

void SDL::Event::addHitToEvent(float x, float y, float z, unsigned int detId)
{
    const int HIT_MAX = 1000000;
    const int HIT_2S_MAX = 100000;

    if(hitsInGPU == nullptr)
    {
        hipMallocManaged(&hitsInGPU, sizeof(SDL::hits));
        createHitsInUnifiedMemory(*hitsInGPU,HIT_MAX,HIT_2S_MAX);
    }

    //calls the addHitToMemory function
    addHitToMemory(*hitsInGPU, *modulesInGPU, x, y, z, detId);

    unsigned int moduleLayer = modulesInGPU->layers[(*detIdToIndex)[detId]];
    unsigned int subdet = modulesInGPU->subdets[(*detIdToIndex)[detId]];

    if(subdet == Barrel)
    {
        n_hits_by_layer_barrel_[moduleLayer-1]++;
    }
    else
    {
        n_hits_by_layer_endcap_[moduleLayer-1]++;
    } 

}

void SDL::Event::addMiniDoubletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(mdsInGPU->nMDs[idx] == 0 or modulesInGPU->hitRanges[idx * 2] == -1)
        {
            modulesInGPU->mdRanges[idx * 2] = -1;
            modulesInGPU->mdRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->mdRanges[idx * 2] = idx * N_MAX_MD_PER_MODULES;
            modulesInGPU->mdRanges[idx * 2 + 1] = (idx * N_MAX_MD_PER_MODULES) + mdsInGPU->nMDs[idx] - 1;
    
            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_minidoublets_by_layer_barrel_[modulesInGPU->layers[idx] -1] += mdsInGPU->nMDs[idx];
            }
            else
            {
                n_minidoublets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += mdsInGPU->nMDs[idx];
            }

        }
    }
}

void SDL::Event::addSegmentsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        if(segmentsInGPU->nSegments[idx] == 0)
        {
            modulesInGPU->segmentRanges[idx * 2] = -1;
            modulesInGPU->segmentRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->segmentRanges[idx * 2] = idx * N_MAX_SEGMENTS_PER_MODULE;
            modulesInGPU->segmentRanges[idx * 2 + 1] = idx * N_MAX_SEGMENTS_PER_MODULE + segmentsInGPU->nSegments[idx] - 1;


            if(modulesInGPU->subdets[idx] == Barrel)
            {
  
                n_segments_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += segmentsInGPU->nSegments[idx];
            }
            else
            {
                n_segments_by_layer_endcap_[modulesInGPU->layers[idx] -1] += segmentsInGPU->nSegments[idx];
            }
        }
    }
}

void SDL::Event::createMiniDoublets()
{
    hipDeviceSynchronize();
    auto memStart = std::chrono::high_resolution_clock::now();
    if(mdsInGPU == nullptr)
    {
        hipMallocManaged(&mdsInGPU, sizeof(SDL::miniDoublets));
    	createMDsInUnifiedMemory(*mdsInGPU, N_MAX_MD_PER_MODULES, nModules);
    }
    hipDeviceSynchronize();
    auto memStop = std::chrono::high_resolution_clock::now();
    auto memDuration = std::chrono::duration_cast<std::chrono::milliseconds>(memStop - memStart); //in milliseconds

    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((nLowerModules % nThreads.x == 0 ? nLowerModules/nThreads.x : nLowerModules/nThreads.x + 1),(N_MAX_HITS_PER_MODULE % nThreads.y == 0 ? N_MAX_HITS_PER_MODULE/nThreads.y : N_MAX_HITS_PER_MODULE/nThreads.y + 1), (N_MAX_HITS_PER_MODULE % nThreads.z == 0 ? N_MAX_HITS_PER_MODULE/nThreads.z : N_MAX_HITS_PER_MODULE/nThreads.z + 1));
    //std::cout<<nBlocks.x<<" "<<nBlocks.y<<" "<<nBlocks.z<<" "<<std::endl;

//    int nThreads = 1;
//    int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    hipDeviceSynchronize();
    auto syncStart = std::chrono::high_resolution_clock::now();

    createMiniDoubletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU,*hitsInGPU,*mdsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    auto syncStop = std::chrono::high_resolution_clock::now();

    auto syncDuration =  std::chrono::duration_cast<std::chrono::milliseconds>(syncStop - syncStart);

    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    
    }
    
    addMiniDoubletsToEvent();


}

void SDL::Event::createSegmentsWithModuleMap()
{
    if(segmentsInGPU == nullptr)
    {
        hipMallocManaged(&segmentsInGPU, sizeof(SDL::segments));
        createSegmentsInUnifiedMemory(*segmentsInGPU, N_MAX_SEGMENTS_PER_MODULE, nModules);
    }
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

//    dim3 nThreads(1,16,16);
//    dim3 nBlocks(((nLowerModules * MAX_CONNECTED_MODULES)  % nThreads.x == 0 ? (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x : (nLowerModules * MAX_CONNECTED_MODULES)/nThreads.x + 1),(N_MAX_MD_PER_MODULES % nThreads.y == 0 ? N_MAX_MD_PER_MODULES/nThreads.y : N_MAX_MD_PER_MODULES/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));

    unsigned int nThreads = 1;   
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createSegmentsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    
    }
    addSegmentsToEvent();

}


void SDL::Event::createTrackletsWithModuleMap()
{
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipMallocManaged(&trackletsInGPU, sizeof(SDL::tracklets));
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
    }

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    

    }
    /*addTrackletsToEvent will be called in the createTrackletsWithAGapWithModuleMap function*/
}


void SDL::Event::createTrackletsWithAGapWithModuleMap()
{
    //use the same trackletsInGPU as before if it exists
    unsigned int nLowerModules = *modulesInGPU->nLowerModules;

    //TRCAKLETS - To conserve memory, we shall be only declaring nLowerModules amount of memory!!!!!!!
    if(trackletsInGPU == nullptr)
    {
        hipMallocManaged(&trackletsInGPU, sizeof(SDL::tracklets));
        createTrackletsInUnifiedMemory(*trackletsInGPU, N_MAX_TRACKLETS_PER_MODULE , nLowerModules);
    }

    unsigned int nThreads = 1;
    unsigned int nBlocks = nLowerModules % nThreads == 0 ? nLowerModules/nThreads : nLowerModules/nThreads + 1;

    createTrackletsWithAGapInGPU<<<nBlocks,nThreads>>>(*modulesInGPU, *hitsInGPU, *mdsInGPU, *segmentsInGPU, *trackletsInGPU);

    hipError_t cudaerr = hipDeviceSynchronize();
    if(cudaerr != hipSuccess)
    {
        std::cout<<"sync failed with error : "<<hipGetErrorString(cudaerr)<<std::endl;    

    }
    addTrackletsToEvent();

}


void SDL::Event::addTrackletsToEvent()
{
    unsigned int idx;
    for(unsigned int i = 0; i<*(SDL::modulesInGPU->nLowerModules); i++)
    {
        idx = SDL::modulesInGPU->lowerModuleIndices[i];
        //tracklets run only on lower modules!!!!!!
        if(trackletsInGPU->nTracklets[i] == 0)
        {
            modulesInGPU->trackletRanges[idx * 2] = -1;
            modulesInGPU->trackletRanges[idx * 2 + 1] = -1;
        }
        else
        {
            modulesInGPU->trackletRanges[idx * 2] = idx * N_MAX_TRACKLETS_PER_MODULE;
            modulesInGPU->trackletRanges[idx * 2 + 1] = idx * N_MAX_TRACKLETS_PER_MODULE + trackletsInGPU->nTracklets[i] - 1;
 
            if(modulesInGPU->subdets[idx] == Barrel)
            {
                n_tracklets_by_layer_barrel_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
            else
            {
                n_tracklets_by_layer_endcap_[modulesInGPU->layers[idx] - 1] += trackletsInGPU->nTracklets[i];
            }
        }
    }
}

__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;
    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;
    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);
    
    if(success)
    {
        unsigned int mdModuleIndex = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        unsigned int mdIndex = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIndex;

        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIndex);
    }
}

/*__global__ void createMiniDoubletsFromLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, unsigned int lowerModuleIndex, unsigned int upperModuleIndex, unsigned int nLowerHits, unsigned int nUpperHits)
{
    unsigned int lowerHitIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int upperHitIndex = blockIdx.z * blockDim.z + threadIdx.z;

    //consider assigining a dummy computation function for these
    if(lowerHitIndex >= nLowerHits) return;
    if(upperHitIndex >= nUpperHits) return;

    unsigned int lowerHitArrayIndex = modulesInGPU.hitRanges[lowerModuleIndex * 2] + lowerHitIndex;
    unsigned int upperHitArrayIndex = modulesInGPU.hitRanges[upperModuleIndex * 2] + upperHitIndex;

    float dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange;

    bool success = runMiniDoubletDefaultAlgo(modulesInGPU, hitsInGPU, lowerModuleIndex, lowerHitArrayIndex, upperHitArrayIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange);

    if(success)
    {
        unsigned int mdModuleIdx = atomicAdd(&mdsInGPU.nMDs[lowerModuleIndex],1);
        unsigned int mdIdx = lowerModuleIndex * N_MAX_MD_PER_MODULES + mdModuleIdx;

        addMDToMemory(mdsInGPU,hitsInGPU, modulesInGPU, lowerHitArrayIndex, upperHitArrayIndex, lowerModuleIndex, dz, dphi, dphichange, shiftedX, shiftedY, shiftedZ, noShiftedDz, noShiftedDphi, noShiftedDphiChange, mdIdx);
    }
}


__global__ void createMiniDoubletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU)
{
    int lowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(lowerModuleArrayIndex >= (*modulesInGPU.nLowerModules)) return; //extra precaution

    int lowerModuleIndex = modulesInGPU.lowerModuleIndices[lowerModuleArrayIndex];
    int upperModuleIndex = modulesInGPU.partnerModuleIndex(lowerModuleIndex);

    if(modulesInGPU.hitRanges[lowerModuleIndex * 2] == -1) return;
    if(modulesInGPU.hitRanges[upperModuleIndex * 2] == -1) return;

    unsigned int nLowerHits = modulesInGPU.hitRanges[lowerModuleIndex * 2 + 1] - modulesInGPU.hitRanges[lowerModuleIndex * 2] + 1;
    unsigned int nUpperHits = modulesInGPU.hitRanges[upperModuleIndex * 2 + 1] - modulesInGPU.hitRanges[upperModuleIndex * 2] + 1;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(1,nLowerHits % nThreads.y == 0 ? nLowerHits/nThreads.y : nLowerHits/nThreads.y + 1, nUpperHits % nThreads.z == 0 ? nUpperHits/nThreads.z : nUpperHits/nThreads.z + 1);

    createMiniDoubletsFromLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, lowerModuleIndex, upperModuleIndex, nLowerHits, nUpperHits);

  
}*/

/*__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int xAxisIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int innerMDArrayIdx = blockIdx.y * blockDim.y + threadIdx.y;
    int outerMDArrayIdx = blockIdx.z * blockDim.z + threadIdx.z;

    int innerLowerModuleArrayIdx = xAxisIdx/MAX_CONNECTED_MODULES;
    int outerLowerModuleArrayIdx = xAxisIdx % MAX_CONNECTED_MODULES; //need this index from the connected module array
    
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIdx];

    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];

    if(outerLowerModuleArrayIdx >= nConnectedModules) return;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIdx];

    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex];
    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex];

    if(innerMDArrayIdx >= nInnerMDs) return;
    if(outerMDArrayIdx >= nOuterMDs) return;

    unsigned int innerMDIndex = modulesInGPU.mdRanges[innerLowerModuleIndex * 2] + innerMDArrayIdx;
    unsigned int outerMDIndex = modulesInGPU.mdRanges[outerLowerModuleIndex * 2] + outerMDArrayIdx;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;

        addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
    }


}*/


__global__ void createSegmentsFromInnerLowerModule(struct SDL::modules&modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, unsigned int innerLowerModuleIndex, unsigned int nInnerMDs)
{
    unsigned int outerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int innerMDArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int outerMDArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    unsigned int outerLowerModuleIndex = modulesInGPU.moduleMap[innerLowerModuleIndex * MAX_CONNECTED_MODULES + outerLowerModuleArrayIndex];

    unsigned int nOuterMDs = mdsInGPU.nMDs[outerLowerModuleIndex];
    if(innerMDArrayIndex >= nInnerMDs) return;
    if(outerMDArrayIndex >= nOuterMDs) return;

    unsigned int innerMDIndex = innerLowerModuleIndex * N_MAX_MD_PER_MODULES + innerMDArrayIndex;
    unsigned int outerMDIndex = outerLowerModuleIndex * N_MAX_MD_PER_MODULES + outerMDArrayIndex;

    float zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD;

    unsigned int innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex;

    dPhiMin = 0;
    dPhiMax = 0;
    dPhiChangeMin = 0;
    dPhiChangeMax = 0;

    bool success = runSegmentDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, innerLowerModuleIndex, outerLowerModuleIndex, innerMDIndex, outerMDIndex, zIn, zOut, rtIn, rtOut, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex);

    if(success)
    {
        unsigned int segmentModuleIdx = atomicAdd(&segmentsInGPU.nSegments[innerLowerModuleIndex],1);
        unsigned int segmentIdx = innerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + segmentModuleIdx;

        addSegmentToMemory(segmentsInGPU,innerMDIndex, outerMDIndex,innerLowerModuleIndex, outerLowerModuleIndex, innerMiniDoubletAnchorHitIndex, outerMiniDoubletAnchorHitIndex, dPhi, dPhiMin, dPhiMax, dPhiChange, dPhiChangeMin, dPhiChangeMax, zIn, zOut, rtIn, rtOut, dAlphaInnerMDSegment, dAlphaOuterMDSegment, dAlphaInnerMDOuterMD,segmentIdx);
    }

}

__global__ void createSegmentsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU)
{
    int innerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerLowerModuleArrayIndex];
    unsigned int nConnectedModules = modulesInGPU.nConnectedModules[innerLowerModuleIndex];
    unsigned int nInnerMDs = mdsInGPU.nMDs[innerLowerModuleIndex];

    if(nConnectedModules == 0) return;

    if(nInnerMDs == 0) return;
    dim3 nThreads(1,16,16);
    dim3 nBlocks((nConnectedModules % nThreads.x == 0 ? nConnectedModules/nThreads.x : nConnectedModules/nThreads.x + 1), (nInnerMDs % nThreads.y == 0 ? nInnerMDs/nThreads.y : nInnerMDs/nThreads.y + 1), (N_MAX_MD_PER_MODULES % nThreads.z == 0 ? N_MAX_MD_PER_MODULES/nThreads.z : N_MAX_MD_PER_MODULES/nThreads.z + 1));
    createSegmentsFromInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerLowerModuleIndex,nInnerMDs);
   
}


__global__ void createTrackletsFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if(innerSegmentArrayIndex >= nInnerSegments) return;
        //outer inner lower module array indices should be obtained from the partner module of the inner segment's outer lower module
    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;


    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];

    //number of possible outer segment inner MD lower modules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }

}

__global__ void createTrackletsInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Proposal 1 : Inner kernel takes care of both loops
    int xAxisIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex =  blockIdx.y * blockDim.y + threadIdx.y;
    int outerSegmentArrayIndex = blockIdx.z * blockDim.z + threadIdx.z;
    
    if(innerSegmentArrayIndex >= nInnerSegments) return;

    int middleLowerModuleArrayIndex = xAxisIndex / MAX_CONNECTED_MODULES;
    int outerInnerLowerModuleArrayIndex = xAxisIndex % MAX_CONNECTED_MODULES;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex]; 
   
    //first check for middle modules
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;

    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    //second check for outerInnerLowerMoules
    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;

    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }
}

__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 1
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,16,16);
    dim3 nBlocks((MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) % nThreads.x  == 0 ? (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x : (MAX_CONNECTED_MODULES * MAX_CONNECTED_MODULES) / nThreads.x + 1 ,nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,N_MAX_SEGMENTS_PER_MODULE % nThreads.z == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.z : N_MAX_SEGMENTS_PER_MODULE/nThreads.z + 1);

    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex,nInnerSegments,innerInnerLowerModuleArrayIndex);

}

/*__global__ void createTrackletsWithAGapFromMiddleLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int middleLowerModuleIndex, unsigned int innerSegmentIndex, unsigned int innerInnerLowerModuleArrayIndex, unsigned int nOuterInnerLowerModules,unsigned int innerOuterLowerModuleIndex)
{
    //Inner kernel of Proposal 2 : Inner kernel does middle->outer modoule mapping
    int outerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int outerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    
    //check for outerInnerLowerModules
    if(outerInnerLowerModuleArrayIndex >= nOuterInnerLowerModules) return;


    unsigned int outerInnerLowerModuleIndex = modulesInGPU.moduleMap[middleLowerModuleIndex * MAX_CONNECTED_MODULES + outerInnerLowerModuleArrayIndex];

    unsigned int nOuterSegments = segmentsInGPU.nSegments[outerInnerLowerModuleIndex];
    if(outerSegmentArrayIndex >= nOuterSegments) return;

    unsigned int outerSegmentIndex = outerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + outerSegmentArrayIndex;

    //for completeness - outerOuterLowerModuleIndex and innerOuterLowerModuleIndex
    unsigned int outerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[outerSegmentIndex];

    //with both segment indices obtained, run the tracklet algorithm
    
   float zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut;
   float betaInCut, betaOutCut, dBetaCut;

   bool success = runTrackletDefaultAlgo(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, innerInnerLowerModuleIndex, innerOuterLowerModuleIndex, outerInnerLowerModuleIndex, outerOuterLowerModuleIndex, innerSegmentIndex, outerSegmentIndex, zOut, rtOut, deltaPhiPos, deltaPhi, betaIn, betaOut, betaInCut, betaOutCut, dBetaCut); //might want to send the other two module indices and the anchor hits also to save memory accesses
   if(success)
   {
        unsigned int trackletModuleIndex = atomicAdd(&trackletsInGPU.nTracklets[innerInnerLowerModuleArrayIndex],1);
        unsigned int trackletIndex = innerInnerLowerModuleArrayIndex * N_MAX_TRACKLETS_PER_MODULE + trackletModuleIndex;

        addTrackletToMemory(trackletsInGPU,innerSegmentIndex,outerSegmentIndex,innerInnerLowerModuleIndex,innerOuterLowerModuleIndex,outerInnerLowerModuleIndex,outerOuterLowerModuleIndex,zOut,rtOut,deltaPhiPos,deltaPhi,betaIn,betaOut,trackletIndex, betaInCut, betaOutCut, dBetaCut);
   }
}

__global__ void createTrackletsWithAGapFromInnerInnerLowerModule(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU, unsigned int innerInnerLowerModuleIndex, unsigned int nInnerSegments, unsigned int innerInnerLowerModuleArrayIndex)
{
    //Middle kernel of Proposal 2 : middle kernel does the inner->middle module mapping

    int middleLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int innerSegmentArrayIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if(innerSegmentArrayIndex >= nInnerSegments) return;

    unsigned int innerSegmentIndex = innerInnerLowerModuleIndex * N_MAX_SEGMENTS_PER_MODULE + innerSegmentArrayIndex;

    //middle lower module - modules that are connected to outer lower module of inner segment
    unsigned int innerOuterLowerModuleIndex = segmentsInGPU.outerLowerModuleIndices[innerSegmentIndex];
    unsigned int nMiddleLowerModules = modulesInGPU.nConnectedModules[innerOuterLowerModuleIndex];
 
    if(middleLowerModuleArrayIndex >= nMiddleLowerModules) return;
 
    unsigned int middleLowerModuleIndex = modulesInGPU.moduleMap[innerOuterLowerModuleIndex * MAX_CONNECTED_MODULES + middleLowerModuleArrayIndex];

    unsigned int nOuterInnerLowerModules = modulesInGPU.nConnectedModules[middleLowerModuleIndex];

    dim3 nThreads(1,32,1);
    dim3 nBlocks(nOuterInnerLowerModules % nThreads.x  == 0 ? nOuterInnerLowerModules / nThreads.x : nOuterInnerLowerModules / nThreads.x + 1 ,N_MAX_SEGMENTS_PER_MODULE % nThreads.y == 0 ? N_MAX_SEGMENTS_PER_MODULE/nThreads.y : N_MAX_SEGMENTS_PER_MODULE/nThreads.y + 1,1);

    createTrackletsWithAGapFromMiddleLowerModule<<<nBlocks,nThreads>>>(modulesInGPU,hitsInGPU,mdsInGPU,segmentsInGPU,trackletsInGPU,innerInnerLowerModuleIndex, middleLowerModuleIndex,innerSegmentIndex,innerInnerLowerModuleArrayIndex,nOuterInnerLowerModules,innerOuterLowerModuleIndex);


}


__global__ void createTrackletsWithAGapInGPU(struct SDL::modules& modulesInGPU, struct SDL::hits& hitsInGPU, struct SDL::miniDoublets& mdsInGPU, struct SDL::segments& segmentsInGPU, struct SDL::tracklets& trackletsInGPU)
{
    //outer kernel for proposal 2
    int innerInnerLowerModuleArrayIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(innerInnerLowerModuleArrayIndex >= *modulesInGPU.nLowerModules) return;
    unsigned int innerInnerLowerModuleIndex = modulesInGPU.lowerModuleIndices[innerInnerLowerModuleArrayIndex];
    unsigned int nInnerSegments = segmentsInGPU.nSegments[innerInnerLowerModuleIndex];
    if(nInnerSegments == 0) return;

    dim3 nThreads(1,1,1);
    dim3 nBlocks(MAX_CONNECTED_MODULES % nThreads.x  == 0 ? MAX_CONNECTED_MODULES / nThreads.x : MAX_CONNECTED_MODULES / nThreads.x + 1 , nInnerSegments % nThreads.y == 0 ? nInnerSegments/nThreads.y : nInnerSegments/nThreads.y + 1,1);
    
    createTrackletsWithAGapFromInnerInnerLowerModule<<<nBlocks,nThreads>>>(modulesInGPU, hitsInGPU, mdsInGPU, segmentsInGPU, trackletsInGPU, innerInnerLowerModuleIndex, nInnerSegments, innerInnerLowerModuleArrayIndex);

}*/



unsigned int SDL::Event::getNumberOfHits()
{
    unsigned int hits = 0;
    for(auto &it:n_hits_by_layer_barrel_)
    {
        hits += it;
    }
    for(auto& it:n_hits_by_layer_endcap_)
    {
        hits += it;
    }

    return hits;
}

unsigned int SDL::Event::getNumberOfHitsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_hits_by_layer_barrel_[layer];
    else
        return n_hits_by_layer_barrel_[layer] + n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerBarrel(unsigned int layer)
{
    return n_hits_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfHitsByLayerEndcap(unsigned int layer)
{
    return n_hits_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoublets()
{
     unsigned int miniDoublets = 0;
    for(auto &it:n_minidoublets_by_layer_barrel_)
    {
        miniDoublets += it;
    }
    for(auto &it:n_minidoublets_by_layer_endcap_)
    {
        miniDoublets += it;
    }

    return miniDoublets;
   
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_minidoublets_by_layer_barrel_[layer];
    else
        return n_minidoublets_by_layer_barrel_[layer] + n_minidoublets_by_layer_endcap_[layer];   
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerBarrel(unsigned int layer)
{
    return n_minidoublets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfMiniDoubletsByLayerEndcap(unsigned int layer)
{
    return n_minidoublets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfSegments()
{
    unsigned int segments = 0;
    for(auto &it:n_segments_by_layer_barrel_)
    {
        segments += it;
    }
    for(auto &it:n_segments_by_layer_endcap_)
    {
        segments += it;
    }

    return segments;
   
}

unsigned int SDL::Event::getNumberOfSegmentsByLayer(unsigned int layer)
{
     if(layer == 6)
        return n_segments_by_layer_barrel_[layer];
    else
        return n_segments_by_layer_barrel_[layer] + n_segments_by_layer_endcap_[layer];   
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerBarrel(unsigned int layer)
{
    return n_segments_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfSegmentsByLayerEndcap(unsigned int layer)
{
    return n_segments_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTracklets()
{
    unsigned int tracklets = 0;
    for(auto &it:n_tracklets_by_layer_barrel_)
    {
        tracklets += it;
    }
    for(auto &it:n_tracklets_by_layer_endcap_)
    {
        tracklets += it;
    }

    return tracklets;
   
}

unsigned int SDL::Event::getNumberOfTrackletsByLayer(unsigned int layer)
{
    if(layer == 6)
        return n_tracklets_by_layer_barrel_[layer];
    else
        return n_tracklets_by_layer_barrel_[layer] + n_tracklets_by_layer_endcap_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerBarrel(unsigned int layer)
{
    return n_tracklets_by_layer_barrel_[layer];
}

unsigned int SDL::Event::getNumberOfTrackletsByLayerEndcap(unsigned int layer)
{
    return n_tracklets_by_layer_endcap_[layer];
}


struct SDL::hits* SDL::Event::getHits()
{
    return hitsInGPU;
}

struct SDL::miniDoublets* SDL::Event::getMiniDoublets()
{
    return mdsInGPU;
}

struct SDL::segments* SDL::Event::getSegments()
{
    return segmentsInGPU;
}

struct SDL::tracklets* SDL::Event::getTracklets()
{
    return trackletsInGPU;
}

